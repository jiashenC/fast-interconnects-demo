
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>

// Add a scalar to the vector
__global__ void vadd(int *const v, int const a, size_t const len) {
  const unsigned int gid = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int gsize = gridDim.x * blockDim.x;

  for (size_t i = gid; i < len; i += gsize) {
    v[i] += a;
  }
}

int main() {
  // Vector length
  constexpr size_t LEN = 100'000;

  // GPU kernel parameters
  constexpr unsigned int grid = 160;
  constexpr unsigned int block = 1024;

  // Allocate vector
  std::cout << "Allocate vector on CPU" << std::endl;
  int *data = nullptr;
  data = reinterpret_cast<int *>(malloc(LEN * sizeof(int)));
  if (data == nullptr) {
    std::cerr << "Failed to allocate memory" << std::endl;
    std::exit(EXIT_FAILURE);
  }

  // Initialize vector with some data
  std::cout << "Init vector on CPU" << std::endl;
  for (size_t i = 0; i < LEN; ++i) {
    data[i] = i;
  }

  // Call a function to do some work
  std::cout << "Process on GPU" << std::endl;
  vadd<<<grid, block>>>(data, 1, LEN);

  // Wait for the GPU kernel to finish execution
  hipDeviceSynchronize();

  // Verify that result is correct
  std::cout << "Get and verify results" << std::endl;
  unsigned long long sum = 0;
  for (size_t i = 0; i < LEN; ++i) {
    sum += data[i];
  }
  assert(sum == (LEN * (LEN + 1)) / 2);

  // Free vector
  std::cout << "Free memory" << std::endl;
  free(data);

  std::exit(EXIT_SUCCESS);
}
